
#include <hip/hip_runtime.h>
extern "C"
__global__ void mul(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
    }

}
